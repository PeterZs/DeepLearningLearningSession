#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_weighted_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithWeightedLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    Forward_cpu(bottom, top);
}

template <typename Dtype>
void SoftmaxWithWeightedLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithWeightedLossLayer);

}  // namespace caffe
